// blocking factor = 64
// blocks = 40000 / 64 = 625 *625
// threads = 32*32

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>


#define INF ((1 << 30) - 1)
// #define V 40010  // 2 ≤ V ≤ 40000 (Single-GPU)
#define B 64  
// 獲取時間戳
double getTimeStamp() {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return (double) tv.tv_sec + (double) tv.tv_usec / 1000000;
}
void input(char *infile);
void output(char *outFileName);
void block_FW();
__global__ void phase1(int r, int *d_Dist, int n);
__global__ void phase2(int r, int *d_Dist, int n);
__global__ void phase3(int r, int *d_Dist, int n);

int *h_Dist;
int *d_Dist;
int n, m, n_original;

int ceil(int a, int b) { return (a + b - 1) / b; }

int main(int argc, char *argv[]) {
    double start, end, io_time = 0.0, output_time = 0.0, compute_time = 0.0, input_time = 0.0,start_time = 0.0f, end_time = 0.0f, total_time = 0.0f;
    start_time = getTimeStamp();
    start = getTimeStamp();
    input(argv[1]);
    end = getTimeStamp();
    input_time += end - start;

    start = getTimeStamp();
    block_FW();
    end = getTimeStamp();
    compute_time += end - start;

    start = getTimeStamp();
    output(argv[2]);
    end = getTimeStamp();
    output_time += end - start;
    io_time = input_time+output_time;
    end_time = getTimeStamp();
    total_time = end_time-start_time;
    // 打印結果
    printf("I/O Input Time: %.2f seconds\n", input_time);
    printf("I/O output Time: %.2f seconds\n", output_time);
    printf("I/O total Time: %.2f seconds\n", io_time);
    printf("Compute Time: %.2f seconds\n", compute_time);
    printf("Total Time: %.2f seconds\n", total_time);       
    return 0;

}
void block_FW(){
    int round = ceil(n/B);      // perform ⌈𝑉/𝐵⌉ rounds
    int shared_memory_size = B * B * sizeof(int);
    dim3 block(32, 32);   // Machcine 上限

    // Allocate memory for d_Dist and copy data from host to device
    hipMalloc(&d_Dist, n * n * sizeof(int));
    hipMemcpy(d_Dist, h_Dist, n * n * sizeof(int), hipMemcpyHostToDevice);

    for (int r = 0; r < round; r++) {
        phase1<<<1, block, shared_memory_size>>>(r, d_Dist, n);
        phase2<<<dim3(2, round - 1), block, 2 * shared_memory_size>>>(r, d_Dist, n);
        phase3<<<dim3(round - 1, round - 1), block, 2 * shared_memory_size>>>(r, d_Dist, n);
    }

    // Copy data from device to host
    hipMemcpy(h_Dist, d_Dist, n * n * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_Dist);

}
__global__ void phase1(int r, int *d_Dist, int n) {
    extern __shared__ int shared_memory[];
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int b_x = r * B;    // 對d_dist來說 Block x y 的起始座標 
    int b_y = r * B;

    // Copy data from global memory to shared memory， Block factor & threads 嚴重影響，
    // 因為在此用B/2位移，但如果B=128 blocksize=32，應為 B/3 ,所以不能套用
    // 4 個 Threads Block 並不會同時載入全部共享記憶體，而是分批次進行。
    shared_memory[ty * B + tx] = d_Dist[(b_y + ty) * n + (b_x + tx)];                           // upper left
    shared_memory[ty * B + (B/2 + tx)] = d_Dist[(b_y + ty) * n + (b_x + B/2 + tx)];             // upper right
    shared_memory[(B/2 + ty) * B + tx] = d_Dist[(b_y + ty + B/2)*n + b_x + tx];                 // Bottom left
    shared_memory[(B/2 + ty) * B + (B/2 + tx)] = d_Dist[(b_y + ty + B/2)*n + (b_x + B/2 + tx)]; // Bottom right
    __syncthreads();

    // share memory 內部運算，更新最短路徑
    // shared_memory[i][j]=min(shared_memory[i][j],shared_memory[i][k]+shared_memory[k][j])
    #pragma unroll
    for(int k = 0; k < B; ++k){
        shared_memory[ty * B + tx] = min(shared_memory[ty * B + tx],
                                         shared_memory[ty * B + k] + shared_memory[k * B + tx]);
        shared_memory[ty * B + (B/2 + tx)] = min(shared_memory[ty * B + (B/2 + tx)],
                                                 shared_memory[ty * B + k] + shared_memory[k*B + (tx + B / 2)]);
        shared_memory[(B/2 + ty) * B + tx] = min(shared_memory[(B/2 + ty) * B + tx], 
                                                 shared_memory[(ty + B / 2) * B + k] + shared_memory[k * B + tx]);
        shared_memory[(B/2 + ty) * B + (B/2 + tx)] = min(shared_memory[(B/2 + ty) * B + (B/2 + tx)],
                                                         shared_memory[(ty + B / 2) * B + k] + shared_memory[k*B + (tx + B / 2)]);                                
    }

    // Copy data from shared memory to global memory
    d_Dist[(b_y + ty) * n + (b_x + tx)] = shared_memory[ty * B + tx];
    d_Dist[(b_y + ty) * n + (b_x + B/2 + tx)] = shared_memory[ty * B + (B/2 + tx)];
    d_Dist[(b_y + ty + B/2)*n + b_x + tx] = shared_memory[(B/2 + ty) * B + tx];
    d_Dist[(b_y + ty + B/2)*n + (b_x + B/2 + tx)] = shared_memory[(B/2 + ty) * B + (B/2 + tx)];
}
__global__ void phase2(int r, int *d_Dist, int n){
    // 在 CUDA 中，dim3 grid(2, round - 1) 決定了 處理的 Block 順序
    // 依序做block(1,0) (1,1) (1,3) (0,0) (0,1) (0,3)每個block跟pivot的計算

    extern __shared__ int shared_memory[];
    int tx = threadIdx.x;
    int ty = threadIdx.y; 
    int bx = blockIdx.x;  // range [0,1]
    int by = blockIdx.y;
    int b_offset = r * B;

    // (!bx) => row or column major, bx = 0 do column major
    // when r = 2, deal with Block(2,0),(2,1),(2,3) respectively
    // b_i 和 b_j 分別代表 CUDA Block 在全域矩陣中的起始座標
    int b_i = (bx * r + (!bx) * (by + (by >= r)) ) * B;
    int b_j = (bx * (by + (by >= r)) + (!bx)*r) * B;

    // Copy data from pivot, registers faster
    int val0 =  d_Dist[(b_i + ty) * n + (b_j + tx)];                
    int val1 =  d_Dist[(b_i + ty) * n + (b_j + tx + B/2)];          
    int val2 =  d_Dist[(b_i + ty + B/2) * n + (b_j + tx)];          
    int val3 =  d_Dist[(b_i + ty + B/2) * n + (b_j + tx + B/2)];    

    // Copy data from global memory to shared memory for row major
    shared_memory[ty*B + tx] =  d_Dist[(b_i + ty) * n + (b_offset + tx)];                                 
    shared_memory[ty*B + (tx + B/2)] =  d_Dist[(b_i + ty) * n + (b_offset + tx + B/2)];                   
    shared_memory[(ty + B/2)*B + tx] =  d_Dist[(b_i + ty + B/2) * n + (b_offset + tx)];                   
    shared_memory[(ty + B/2)*B + (tx + B/2)] =  d_Dist[(b_i + ty + B/2) * n + (b_offset + tx + B/2)];   

    // Copy data from global memory to shared memory for column major
    shared_memory[B*B + ty*B + tx] =  d_Dist[(b_offset + ty) * n + (b_j + tx)];                               
    shared_memory[B*B + ty*B + (tx + B/2)] =  d_Dist[(b_offset + ty) * n + (b_j + tx + B/2)];                 
    shared_memory[B*B + (ty + B/2)*B + tx] =  d_Dist[(b_offset + ty + B/2) * n + (b_j + tx)];                 
    shared_memory[B*B + (ty + B/2)*B + (tx + B/2)] =  d_Dist[(b_offset + ty + B/2) * n + (b_j + tx + B/2)];   
    __syncthreads();

    #pragma unroll
    for (int k = 0; k < B; ++k) {
        val0 =  min(val0, shared_memory[ty*B + k]+ shared_memory[B*B + k*B + tx]);                 //Block upper left
        val1 =  min(val1, shared_memory[ty*B + k]+ shared_memory[B*B + k*B + (tx + B/2)]);          //Block upper right
        val2 =  min(val2, shared_memory[(ty + B/2)*B + k]+ shared_memory[B*B + k*B + tx]);          //Block upper left
        val3 =  min(val3, shared_memory[(ty + B/2)*B + k]+ shared_memory[B*B + k*B + (tx + B/2)]);   //Block upper left
    }    

    // Copy data from shared memory to global memory
    d_Dist[(b_i + ty) * n + (b_j + tx)] = val0;              
    d_Dist[(b_i + ty) * n + (b_j + tx + B/2)] = val1;        
    d_Dist[(b_i + ty + B/2) * n + (b_j + tx)] = val2 ;       
    d_Dist[(b_i + ty + B/2) * n + (b_j + tx + B/2)] = val3;  

}
__global__ void phase3(int r, int *d_Dist, int n){
    extern __shared__ int shared_memory[];
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int b_i = (blockIdx.x + (blockIdx.x >= r)) * B;
    int b_j = (blockIdx.y + (blockIdx.y >= r)) * B;
    int b_offset = r * B;    

    // Data from current Block, not pivot
    int val0 =  d_Dist[(b_i + ty) * n + (b_j + tx)];                
    int val1 =  d_Dist[(b_i + ty) * n + (b_j + tx + B/2)];          
    int val2 =  d_Dist[(b_i + ty + B/2) * n + (b_j + tx)];          
    int val3 =  d_Dist[(b_i + ty + B/2) * n + (b_j + tx + B/2)]; 

    // Copy data from global memory to shared memory for row major from phase 2
    shared_memory[ty*B + tx] =  d_Dist[(b_i + ty) * n + (b_offset + tx)];                                 //Block upper left
    shared_memory[ty*B + (tx + B/2)] =  d_Dist[(b_i + ty) * n + (b_offset + tx + B/2)];                   //Block upper right
    shared_memory[(ty + B/2)*B + tx] =  d_Dist[(b_i + ty + B/2) * n + (b_offset + tx)];                   //Block upper left
    shared_memory[(ty + B/2)*B + (tx + B/2)] =  d_Dist[(b_i + ty + B/2) * n + (b_offset + tx + B/2)];     //Block upper left
    
    // Copy data from global memory to shared memory for column major from phase 2
    shared_memory[B*B + ty*B + tx] =  d_Dist[(b_offset + ty) * n + (b_j + tx)];                        //Block upper left
    shared_memory[B*B + ty*B + (tx + B/2)] =  d_Dist[(b_offset + ty) * n + (b_j + tx + B/2)];          //Block upper right
    shared_memory[B*B + (ty + B/2)*B + tx] =  d_Dist[(b_offset + ty + B/2) * n + (b_j + tx)];          //Block upper left
    shared_memory[B*B + (ty + B/2)*B + (tx + B/2)] =  d_Dist[(b_offset + ty + B/2) * n + (b_j + tx + B/2)];    //Block upper left
    __syncthreads();

    #pragma unroll
    for (int k = 0; k < B; ++k) {
        val0 =  min(val0, shared_memory[ty*B + k]+ shared_memory[B*B + k*B + tx]);                 //Block upper left
        val1 =  min(val1, shared_memory[ty*B + k]+ shared_memory[B*B + k*B + (tx + B/2)]);          //Block upper right
        val2 =  min(val2, shared_memory[(ty + B/2)*B + k]+ shared_memory[B*B + k*B + tx]);          //Block upper left
        val3 =  min(val3, shared_memory[(ty + B/2)*B + k]+ shared_memory[B*B + k*B + (tx + B/2)]);   //Block upper left
    }

    d_Dist[(b_i + ty) * n + (b_j + tx)] = val0;             
    d_Dist[(b_i + ty) * n + (b_j + tx + B/2)] = val1;       
    d_Dist[(b_i + ty + B/2) * n + (b_j + tx)] = val2 ;       
    d_Dist[(b_i + ty + B/2) * n + (b_j + tx + B/2)] = val3;
}

void input(char *infile) {
    FILE *file = fopen(infile, "rb");
    fseek(file, 0, SEEK_END);
    long file_size = ftell(file);
    rewind(file);

    char *buffer = (char *)malloc(file_size);
    fread(buffer, 1, file_size, file);
    fclose(file);

    int *ptr = (int *)buffer;
    n = *ptr++;
    m = *ptr++;

    n_original = n;
    n += B - ((n % B + B - 1) % B + 1);

    // Allocate pinned memory for h_Dist
    hipHostMalloc(&h_Dist, n * n * sizeof(int), hipHostMallocDefault);

    // Initialize h_Dist in parallel
    #pragma omp parallel for collapse(2)
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            if (i < n_original && j < n_original) {
                h_Dist[i * n + j] = (i == j) ? 0 : INF;
            } else {
                h_Dist[i * n + j] = INF;
            }
        }
    }

    // Parse edges directly from buffer
    int *edges = ptr;
    #pragma omp parallel for
    for (int i = 0; i < m; ++i) {
        int u = edges[i * 3];
        int v = edges[i * 3 + 1];
        int w = edges[i * 3 + 2];
        h_Dist[u * n + v] = w;
    }

    free(buffer);
}

void output(char *outFileName) {
    FILE *file = fopen(outFileName, "wb");

    // 先將 h_Dist 的內容寫入到一個暫存緩衝區中，再一次性將該緩衝區的所有內容寫入文件，這樣可以大幅減少 I/O 的次數，提高輸出速度。
    int *output_buffer = (int *)malloc(n_original * n_original * sizeof(int));

    // 填充 output_buffer
    #pragma omp parallel for
    for (int i = 0; i < n_original; i++) {
        for (int j = 0; j < n_original; j++) {
            int dist_value = h_Dist[i * n + j];
            if (dist_value >= INF) {
                dist_value = 1073741823;
            }
            output_buffer[i * n_original + j] = dist_value;
        }
    }

    fwrite(output_buffer, sizeof(int), n_original * n_original, file);

    fclose(file);
    free(output_buffer);
    hipHostFree(h_Dist);
}

